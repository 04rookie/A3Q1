#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
__global__ void gkernel(int n, float h, float* x, float* y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        const double d = 1.0 / sqrtf(2 * (HIP_PI));
        double sum = 0.0;
        double k = 0.0;
        for(int j = 0; j < n; j++){
            k = (x[i] - x[j])/h;
            sum += d * __expf(-(k * k) / 2);
        }
        y[i] = (1.0/(n*h)) * sum;
    }
    return;
}

__host__ void run_gkernel(int n, float h, float* x, float* y){
    int block_size = 1024;
    int num_blocks = (n + block_size -1)/block_size;
    gkernel<<<num_blocks, block_size>>>(n, h, x, y);
}